#include <stdio.h>
#include <hip/hip_runtime.h>
 

extern "C" {
    int maxmul() {
      size_t free_t, total_t;
      hipMemGetInfo(&free_t, &total_t);
      return int(free_t/1024/1024);
    };

}
